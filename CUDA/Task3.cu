/*******************************
*** *** TASK-3
*** NAME: - SOAIBUZZAMAN
*** Matrikel Number: 613488
*********************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int N = 200;
const int block_size = 32;
const int num_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);

// Device Function
__global__ void calc_max_device(int *vec, int N, int *grid_results)
{
    // Each thread determines the local maximum maxT of its assigned vector elements.
    int max_t = 0;
    __shared__ int part_prod[block_size];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid; i < N; i += blockDim.x * gridDim.x)
    {
        // Max Logic
        if (vec[i] < vec[tid * block_size + i])
        {
            max_t = vec[tid * block_size + i];
        }
    }
    part_prod[threadIdx.x] = max_t;
    __syncthreads();

    // The threads of a blockblockIdx.x determine one local maximum maxB
    // for this block inparallel. This value is stored at the position
    // grid_results[blockIdx.x].
    int size = blockDim.x / 2;
    while (size > 0)
    {
        if (threadIdx.x < size)
        {
            // Finding max for blocksize/2 elements
            if (part_prod[threadIdx.x] < part_prod[threadIdx.x + 1])
                part_prod[threadIdx.x] = part_prod[threadIdx.x + 1];
        }

        __syncthreads();
        size = size / 2;
    }
    // One elements per block
    if (threadIdx.x == 0)
    {
        grid_results[blockIdx.x] = part_prod[0];
    }
}

// Host function
int calc_max(int *vec, int N, int block_size, int num_blocks)
{
    int *max_v;
    int *vec_d, *max_val_d;

    max_v = (int *)malloc(sizeof(int));

    hipMalloc((void **)&vec_d, sizeof(int) * N);
    hipMalloc((void **)&max_val_d, sizeof(int));

    // initialize the vector for testing.
    for (int i = 0; i < N; i++)
    {
        vec[i] = rand() % 100;
    }

    hipMemcpy(vec_d, vec, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(max_val_d, max_v, sizeof(int) * N, hipMemcpyHostToDevice);

    // calling the device funtion
    calc_max_device<<<num_blocks, block_size>>>(vec_d, N, max_val_d);

    hipMemcpy(max_v, max_val_d, sizeof(int), hipMemcpyDeviceToHost);

    free(vec);
    // Couldn't free max_v because it need to be returned.
    hipFree(vec_d);
    hipFree(max_val_d);

    return *max_v;
}

int main(void)
{
    // Main function for testing
    int *vec_h, *max_val_h;

    max_val_h = (int *)malloc(sizeof(int));
    vec_h = (int *)malloc(sizeof(int) * N);

    *max_val_h = calc_max(vec_h, N, block_size, num_blocks);

    printf("%d\n", *max_val_h);

    free(max_val_h);

    return 0;
}
